#include "hip/hip_runtime.h"
#include "rotate_image_cuda.hpp"
#include <string>

#include "cuda_errors.hpp"
#define DIV_UP(x, y) ( (y) * ( ((x)+(y)-1) / (y) ) )
//to hide global texture objects

using namespace cimg_library;
// Texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;

template<typename T>
__global__ void transformKernel(T* outputData,
                                int width,
                                int height,
                                T theta)
{
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    T u = (T)x - (T)width/2;
    T v = (T)y - (T)height/2;
    T tu = u*cosf(theta) - v*sinf(theta);
    T tv = v*cosf(theta) + u*sinf(theta);

    tu /= (T)width;
    tv /= (T)height;

    // read from texture and write to global memory
    T val = tex2D(tex, tu + 0.5f, tv + 0.5f);

    outputData[y*width + x] = val;
}



CImg<float> rotate_cuda(const std::string& filename,
                        const float angle,
                        hipTextureFilterMode filterMode,
                        hipTextureAddressMode addressMode,
                        int normalization)
{
    typedef float T;
    CImg<T> image(filename.c_str());

    T* d = image.data();
    unsigned int width = image.width();
    unsigned int height = image.height();

    size_t size = width * height * sizeof(T);
    T* dd;
    size_t pitch;
    hipMallocPitch((void**)&dd, &pitch, width*sizeof(T), height);
    hipMemcpy2D(dd, pitch, d, width*sizeof(T), width*sizeof(T), height, hipMemcpyHostToDevice);

//    {
//        int nDevices;
//        hipGetDeviceCount(&nDevices);

//        hipDeviceProp_t prop;
//        hipGetDeviceProperties(&prop, nDevices);


//        size_t pitch_size =
//                DIV_UP(width*sizeof(T), prop.textureAlignment)
//                * prop.textureAlignment;

//        std::cout << "PITCH " << pitch
//                  << " MY PITCH: " << pitch_size << std::endl;

//    }


    //prepare texture and allocate image on device;
    hipChannelFormatDesc channelDesc =
            hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray *cuArray;
    cudaSafeCall(hipMallocArray(&cuArray, &channelDesc, width, height));
    cudaSafeCall(hipMemcpy2DToArray(cuArray, 0, 0, dd, pitch, width*sizeof(T), height, hipMemcpyDeviceToDevice));

    tex.addressMode[0] = addressMode;
    tex.addressMode[1] = addressMode;
    tex.filterMode = filterMode;
    tex.normalized = normalization;

    // Bind the array to the texture
    cudaSafeCall(hipBindTextureToArray(tex, cuArray, channelDesc));

    // result data
    T* d_data;
    cudaSafeCall(hipMalloc((void**)&d_data, size));

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    transformKernel<float><<<dimGrid, dimBlock, 0>>>(d_data, width, height, angle);
    cudaCheckError();
    cudaSafeCall(hipDeviceSynchronize());

    //TIMING
    hipEvent_t start, stop;
    cudaSafeCall(hipEventCreate(&start));
    cudaSafeCall(hipEventCreate(&stop));

    const int NTimes = 100;
    hipEventRecord(start);
    for (int i = 0; i < NTimes; i++)
    {
        transformKernel<float><<<dimGrid, dimBlock, 0>>>(d_data, width, height, angle);
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float miliseconds = 0;
    hipEventElapsedTime(&miliseconds, start, stop);
    std::cout << "CUDA TIME: " << miliseconds / (float)NTimes << " ms" <<std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);


    //get back the results on host.
    cudaSafeCall(hipMemcpy(d, d_data, size, hipMemcpyDeviceToHost));

    cudaSafeCall(hipUnbindTexture(tex));

    cudaSafeCall(hipFree(d_data));
    cudaSafeCall(hipFreeArray(cuArray));

    image.save("data/cuda_result.pgm");

    return image;
}

