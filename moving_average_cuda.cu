#include "hip/hip_runtime.h"
#include "moving_average_cuda.hpp"
#include "cuda_errors.hpp"

inline int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

texture<float, 1, hipReadModeElementType> tex1d;
__global__ void moving_average_kernel(float* __restrict__ dst, const int N, const int R)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < N) {

        float average = 0.f;

        for (int k = -R; k <= R; k++) {
            average = average + tex1D(tex1d, (float)(tid - k + 0.5f)/(float)N);
        }

        dst[tid] = average / (2.f * (float)R + 1.f);
    }
}

void moving_average_gpu(float* dst, float* src, const int N, const int R,
                        hipTextureFilterMode filterMode,
                        hipTextureAddressMode addressMode,
                        int normalization)
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    //hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

    hipArray* cuArray;
    cudaSafeCall(hipMallocArray(&cuArray, &channelDesc, N, 1));
    cudaSafeCall(hipMemcpyToArray(cuArray, 0, 0, src, N * sizeof(float), hipMemcpyHostToDevice));

    cudaSafeCall(hipBindTextureToArray(tex1d, cuArray));

    tex1d.filterMode = filterMode;
    tex1d.normalized = normalization;
    //only with normalized!
    tex1d.addressMode[0] = addressMode;

    float* device_result;
    cudaSafeCall(hipMalloc((void**)&device_result, N * sizeof(float)));

    moving_average_kernel<<<iDivUp(N, 256), 256>>>(device_result, N, R);
    cudaCheckError();
    cudaSafeCall(hipDeviceSynchronize());

    cudaSafeCall(hipMemcpy(dst, device_result, N * sizeof(float), hipMemcpyDeviceToHost));

    cudaSafeCall(hipUnbindTexture(tex1d));
    cudaSafeCall(hipFreeArray(cuArray));
    cudaSafeCall(hipFree(device_result));

}

